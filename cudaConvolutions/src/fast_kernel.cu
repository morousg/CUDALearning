#include "hip/hip_runtime.h"
#include "fast_kernel.h"

template <typename O>
__device__ O operate(O i_data){
    return i_data;
}

template <typename I, typename O, typename I2, typename Operation, typename... operations>
__device__ O operate(I i_data, binary_operation_scalar<Operation, I, I2, O> op, operations... ops) {
    O temp = op.nv_operator(i_data, op.scalar);
    return operate(temp, ops...);
}

template <typename I, typename O, typename I2, typename Operation, typename... operations>
__device__ O operate(I i_data, binary_operation_pointer<Operation, I, I2, O> op, operations... ops) {
    // we want to have access to I2 in order to ask for the type size for optimizing
    O temp = op.nv_operator(i_data, op.pointer[GLOBAL_ID]);
    return operate(temp, ops...);
}

template <typename O>
__device__ O operate_optimized(int i, O i_data){
    return i_data;
}

template <typename I, typename O, typename I2, typename Operation, typename... operations>
__device__ O operate_optimized(int i, I i_data, binary_operation_scalar<Operation, I, I2, O> op, operations... ops) {
    O temp = op.nv_operator(i_data, op.scalar);
    return operate_optimized(i, temp, ops...);
}

template <typename I, typename O, typename I2, typename Operation, typename... operations>
__device__ O operate_optimized(int i, I i_data, binary_operation_pointer<Operation, I, I2, O> op, operations... ops) {
    // we want to have access to I2 in order to ask for the type size for optimizing
    O temp = op.nv_operator(i_data, op.temp_register[i]);
    return operate_optimized(i, temp, ops...);
}

template<typename I, typename O, typename... operations>
__global__ void cuda_transform(int size, I* i_data, O* o_data, operations... ops) {
    if (GLOBAL_ID < size) o_data[GLOBAL_ID] = operate(i_data[GLOBAL_ID], ops...);
}

void test_mult_sum_div_float(float* data, dim3 data_dims, hipStream_t stream) {
    // We don't think about step or ROI's yet.
    dim3 thread_block(512);
    dim3 grid(data_dims.x/512);

    binary_operation_scalar<binary_mul<float>, float> op1 = {5.f};
    binary_operation_pointer<binary_sum<float>, float> op2 = {data};
    binary_operation_scalar<binary_div<float>, float> op3 = {2.f};
    binary_operation_scalar<binary_mul<float>, float> op4 = {5.f};
    binary_operation_scalar<binary_div<float>, float> op5 = {3.f};
    binary_operation_scalar<binary_mul<float>, float> op6 = {7.f};

    cuda_transform<<<grid, thread_block, 0, stream>>>(data_dims.x, data, data, op1, op2, op3, op4, op5, op6);
    gpuErrchk(hipGetLastError());
}


// As a first optimization, let's suppose we are always using 4byte types, and we read 4 of them per thread.
// Later on we will play with type sizes and so on.
__device__ void parameter_pointer_read() {}

template <typename I, typename O, typename I2, typename Operation, typename... operations>
__device__ void parameter_pointer_read(binary_operation_pointer<Operation, I, I2, O>& op, operations... ops) {
    uint4* temp = (uint4*)(op.pointer);
    uint4 temp_r = temp[GLOBAL_ID];

    I2 temp0, temp1, temp2, temp3;
    temp0 = ((I2*)(&temp_r))[0];
    temp1 = ((I2*)(&temp_r))[1];
    temp2 = ((I2*)(&temp_r))[2];
    temp3 = ((I2*)(&temp_r))[3];

    op.temp_register[0] = temp0;
    op.temp_register[1] = temp1;
    op.temp_register[2] = temp2;
    op.temp_register[3] = temp3;
}

template <typename I, typename O, typename I2, typename Operation, typename... operations>
__device__ void parameter_pointer_read(binary_operation_scalar<Operation, I, I2, O>& op, operations... ops) {
    parameter_pointer_read(ops...);
}

template<typename I, typename O, typename... operations>
__global__ void cuda_transform_optimized(int size, I* i_data, O* o_data, operations... ops) {

    if (GLOBAL_ID < size) {

        parameter_pointer_read(ops...);

        uint4* i_temp = (uint4*)(i_data);
        uint4 i_temp_r = i_temp[GLOBAL_ID];

        I i_temp0, i_temp1, i_temp2, i_temp3;
        i_temp0 = *((I*)(&i_temp_r.x));
        i_temp1 = *((I*)(&i_temp_r.y));
        i_temp2 = *((I*)(&i_temp_r.z));
        i_temp3 = *((I*)(&i_temp_r.w));

        O res0 = operate_optimized(0, i_temp0, ops...);
        O res1 = operate_optimized(1, i_temp1, ops...);
        O res2 = operate_optimized(2, i_temp2, ops...);
        O res3 = operate_optimized(3, i_temp3, ops...);

        uint4* o_temp = (uint4*)(o_data);
        o_temp[GLOBAL_ID] = make_uint4(*((uint*)&res0), *((uint*)&res1), *((uint*)&res2), *((uint*)&res3));
    }
}

void test_cuda_transform_optimized(float* data, dim3 data_dims, hipStream_t stream) {

     // We don't think about step or ROI's yet.
    dim3 thread_block(512);
    dim3 grid((data_dims.x/512)/4);

    binary_operation_scalar<binary_mul<float>, float> op1 = {5.f};
    binary_operation_pointer<binary_sum<float>, float> op2 = {data};
    binary_operation_scalar<binary_div<float>, float> op3 = {2.f};
    binary_operation_scalar<binary_mul<float>, float> op4 = {5.f};
    binary_operation_scalar<binary_div<float>, float> op5 = {3.f};
    binary_operation_scalar<binary_mul<float>, float> op6 = {7.f};

    cuda_transform_optimized<<<grid, thread_block, 0, stream>>>(data_dims.x, data, data, op1, op2, op3, op4, op5, op6);
    gpuErrchk(hipGetLastError());

}

#include "hip/hip_runtime.h"
#include "kernel.h"
#include <math.h>

#define GLOBAL_ID ( (((blockDim.y * blockIdx.y) + threadIdx.y) * (gridDim.x * blockDim.x)) + ((blockDim.x * blockIdx.x) + threadIdx.x) )

__global__ void basic_copy_kernel(int* datain, int* dataout, int data_size) {
	int global_id = GLOBAL_ID;

	if (global_id < data_size)
		dataout[global_id] = datain[global_id];
}

// limited to data multiple of 4
__global__ void copy_kernel(int* datain, int* dataout, int data_size) {
	int global_id = GLOBAL_ID;

	if (global_id * 4 < data_size) {
		int4* datain_big = (int4*)datain;
		int4* dataout_big = (int4*)dataout;

		dataout[global_id] = datain[global_id];
	}
}

void execute_basic_copy_kernel(int* datain, int* dataout, int data_size, int block_size, hipStream_t stream) {

	int num_blocks = (int)ceilf((float)data_size / (float)block_size);

	basic_copy_kernel<<<num_blocks, block_size, 0, stream>>>(datain, dataout, data_size);
}

void execute_copy_kernel(int* datain, int* dataout, int data_size, int block_size, hipStream_t stream) {

	int num_blocks = (int)ceilf((float)data_size / (float)block_size / 4);

	copy_kernel<<<num_blocks, block_size, 0, stream >>>(datain, dataout, data_size);
}